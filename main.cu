#include <stdio.h>
#include <stdint.h>
#include "nn.cu"

int main(void){

	const int TRAINING_SIZE = 5;
	const int TRAINING_DIM = 8;
	const int L1_SIZE = 8;

	// X, the first 4 lines from Iris dataset
	float h_X[TRAINING_SIZE*TRAINING_DIM] = {
6,148,72,35,0,33.6,0.627,50,
1,85,66,29,0,26.6,0.351,31,
8,183,64,0,0,23.3,0.672,32,
1,89,66,23,94,28.1,0.167,21,
0,137,40,35,168,43.1,2.288,33};

	const signed int X_size = sizeof(h_X);

	float *d_X;
	hipMalloc(&d_X, X_size);
	hipMemcpy(d_X, h_X, X_size, hipMemcpyHostToDevice);

	//WEIGHTS_0
	const long signed int W0_size = L1_SIZE*TRAINING_DIM*sizeof(float);
	float *h_W0 = (float*)malloc(W0_size);
	for (int i = 0; i < L1_SIZE*TRAINING_DIM; i++){
	    h_W0[i] = 0.1 * (2.0*rand()/RAND_MAX-1.0);
	}

	float *d_W0;
	hipMalloc(&d_W0, W0_size);
	hipMemcpy(d_W0, h_W0, W0_size, hipMemcpyHostToDevice);

	//LAYER 1, LAYER 1 DELTA AND BUFFER OF LAYER 1 SIZE
	const long signed int L1_size = L1_SIZE*TRAINING_SIZE*sizeof(float);

	float* h_layer_1 = (float*)malloc(L1_size);
	float* h_layer_1_delta = (float*)malloc(L1_size);
	float* h_buffer = (float*)malloc(L1_size);

	for (int i = 0; i < L1_SIZE*TRAINING_SIZE; i++){
	    h_layer_1[i] = 0.0;
	    h_buffer[i] = 0.0;
	    h_layer_1_delta[i] = 0.0;
	}

	float *d_layer_1;
	hipMalloc(&d_layer_1, L1_size);
	hipMemcpy(d_layer_1, h_layer_1, L1_size, hipMemcpyHostToDevice);

	float *d_buffer;
	hipMalloc(&d_buffer, L1_size);
	hipMemcpy(d_buffer, h_buffer, L1_size, hipMemcpyHostToDevice);

	float *d_layer_1_delta;
	hipMalloc(&d_layer_1_delta, L1_size);
	hipMemcpy(d_layer_1_delta, h_layer_1_delta, L1_size, hipMemcpyHostToDevice);

	//WEIGHTS 1
	const long signed int W1_size = L1_SIZE*sizeof(float);
	float *h_W1 = (float*)malloc(W1_size);
	for (int i = 0; i < L1_SIZE; i++){
	    h_W1[i] = 0.1* (2.0*rand()/RAND_MAX-1.0);
	}

	float *d_W1;
	hipMalloc(&d_W1, W1_size);
	hipMemcpy(d_W1, h_W1, W1_size, hipMemcpyHostToDevice);

	//Y
	float h_y[4] = {	1,
						0,
						1,
						0,
						1};

	const signed int y_size = sizeof(h_y);
	float *d_y;
	hipMalloc(&d_y, y_size);
	hipMemcpy(d_y, h_y, y_size, hipMemcpyHostToDevice);

	//PRED AND PRED_DELTA
	float* h_pred = (float*)malloc(y_size);
	float* h_pred_delta = (float*)malloc(y_size);
	for (int i = 0; i < TRAINING_SIZE; i++){
	    h_pred[i] = 0.0;
	    h_pred_delta[i] = 0.0;
	}

	float *d_pred;
	hipMalloc(&d_pred, y_size);
	hipMemcpy(d_pred, h_pred, y_size, hipMemcpyHostToDevice);

	float *d_pred_delta;
	hipMalloc(&d_pred_delta, y_size);
	hipMemcpy(d_pred_delta, h_pred_delta, y_size, hipMemcpyHostToDevice);

	kfit <<< 1, 1 >>> (	d_X, TRAINING_DIM, TRAINING_SIZE,
						d_y, 1,
						d_layer_1, L1_SIZE, d_layer_1_delta,
						d_pred,
						d_pred_delta,
						d_W0,
						d_W1,
						d_buffer);

	hipMemcpy(h_pred, d_pred, y_size, hipMemcpyDeviceToHost);

	hipFree(d_pred);
	hipFree(d_X);
	hipFree(d_y);
	hipFree(d_layer_1_delta);
	hipFree(d_pred_delta);
	hipFree(d_W0);
	hipFree(d_W1);
	hipFree(d_buffer);

	free(h_layer_1_delta);
	free(h_pred_delta);
	free(h_W0);
	free(h_W1);
	free(h_buffer);

	for (int i = 0; i < TRAINING_SIZE; i++){
		printf("Prediction[%i] : %f True Value[%i] : %f Error[%i] : %f\n", i, h_pred[i], i, h_y[i], i, h_pred[i] - h_y[i]);
	}

	free(h_pred);
}